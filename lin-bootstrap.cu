#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <gsl/gsl_cdf.h> // for calculating std. normal prob,inverse
#include "mystats.h"

// gpu random functions includes
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <unistd.h> // for time(NULL) call

#define MAX_SAMPLE 100

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

float my_ahat(float xin[], int n);// calculate "a" for BCa acceleration
void jack_knife(float xin[], float yin[], int n, float jack_theta[], 
	        float (*func)(float *, float *, int));
float jack_knife_wrapper_slope(float xin[], float yin[], int n);
	       
__host__ __device__ void calc_BLUE_slope_intercept(float xin[], float yin[], int n, 
			      float *slope_out, float *intercept_out);

// this GPU kernel function is used to initialize the random states 
// source: http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html (accessed: 6/5/2017)
__global__ void init_rand_kernel(unsigned int seed, hiprandState_t* states) {
  hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

// main GPU kernel
__global__ void mc_bs_slope_kernel(hiprandState_t* states, float d_x[], 
                            float d_y[], int n, int B, float d_slope[]){
					
	int idx_glob,i,elt_idx,ranval;
	float boot_x[MAX_SAMPLE],boot_y[MAX_SAMPLE];
	float slope,intercept;

	int tot_threads=blockDim.x*gridDim.x;

	// global index corresponds to bootstrap iterate
	idx_glob = blockIdx.x*blockDim.x+threadIdx.x;

	for(elt_idx=idx_glob;elt_idx<B;elt_idx+=tot_threads){
		// randomly sample, store boot populations
		for(i=0;i<n;i++){
			ranval=hiprand(&states[blockIdx.x])%n;
			boot_x[i]=d_x[ranval];
			boot_y[i]=d_y[ranval];
		}
		// calculate BLUE slope and intercept
		calc_BLUE_slope_intercept(boot_x,boot_y,n,&slope,&intercept);
		// store results
		d_slope[elt_idx]=slope;
	}
}
// run full bootstrap!
__global__ void full_bs_slope_kernel(float d_x[], float d_y[], int n, int B,  
                                     float d_slope[]){
					
	int idx_glob,i,elt_idx,pop_idx,divided;
	unsigned long long int skip;
	float boot_x[MAX_SAMPLE],boot_y[MAX_SAMPLE];
	float slope,intercept;

	int tot_threads=blockDim.x*gridDim.x;

	// global index corresponds to bootstrap iterate
	idx_glob = blockIdx.x*blockDim.x+threadIdx.x;
	
	skip=pow(n,n)-1;
	skip/=(n-1);

	for(elt_idx=idx_glob;elt_idx<B;elt_idx+=tot_threads){
		// we skip over guaranteed singularities 
		divided = (elt_idx+1)+(elt_idx+1)*(skip+1)/skip/skip;

		// Grab a bootstrap sample (not random!)
		for(i=0;i<n;i++){
			pop_idx = divided%n;
			boot_x[i]=d_x[pop_idx];
			boot_y[i]=d_y[pop_idx];
			divided/=n;
		}
		// calculate BLUE slope and intercept
		calc_BLUE_slope_intercept(boot_x,boot_y,n,&slope,&intercept);
		// store results
		d_slope[elt_idx]=slope;
	}
}

int main(int argc, char *argv[]){
	int Nbs;
	int thds_per_block = (1<<8);
	int num_blocks = (1<<12);
	int i,npts=0;
	
	int hist_bins=100;
	int hist_counts[hist_bins],bin;
	float hist_max[hist_bins],bin_width;
	
	float alpha,mean,tmp,SE_median,SE_lower,SE_upper;
	float *h_x,*d_x,*h_y,*d_y,*h_slope,*d_slope;
	float lower_percentile,upper_percentile,middle,slope,intercept;
	float BCa_alpha1,BCa_alpha2,z0,lower_BCa,upper_BCa,p_bias;
	float z_lower,z_upper;
	float *jack_knife_array,ahat;
	
	FILE * ipt_fptr;
	FILE * opt_fptr;
	
	char ipt_fname[255];
	char opt_fname[255];
	char readin[255];
	
	hiprandState_t* states;
	float exectime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// input checking
	if(argc!=3){
		printf("Usage: $ %s <filein> <alpha>\n",argv[0]);
		return -1;
	}
	
	sprintf(ipt_fname,"%s",argv[1]);
	
	alpha=strtod(argv[2],NULL);

	//read in data
	ipt_fptr = fopen(ipt_fname,"r");
	if(ipt_fptr==NULL){
		printf("Could not open file %s!\n",ipt_fname);
		return(-07071);
	}
	// count number of points we will be reading
	while(fgets(readin,255,ipt_fptr) != NULL){
		npts++;
	}
	printf("Number of points read in: %d\n",npts);
	rewind(ipt_fptr);
	
	if(npts<3){
		printf("Not enough data to work with, exiting!\n");
		return 0;
	}
	
	// if the power set has less than a million, use full BS (no MC)
	printf("The power set is");
	if(npts<8){
		Nbs = pow(npts,npts)-npts;
		printf(" not too large. Running full Bootstrap.");
	}else{
		Nbs = (1<<20);
		printf(" too large. Running Monte-Carlo Bootstrap.");
	}
	printf("\nThere will be %d BS iterations.\n",Nbs);
	
	// malloc data
	// allocate bootstrap results array
	h_slope=(typeof(h_slope))malloc(Nbs*sizeof(*h_slope));
	if(h_slope==NULL){
		printf("Could not allocate host bootstrap memory in %s!\n",argv[0]);
		return -314;
	}
	// host values, read in from file
	h_x=(typeof(h_x))malloc(npts*sizeof(*h_x));
	h_y=(typeof(h_y))malloc(npts*sizeof(*h_y));
	// jack_knife_array used in BCa
	jack_knife_array=(typeof(jack_knife_array))malloc(npts*sizeof(*jack_knife_array));
	if(h_x==NULL || h_y==NULL || jack_knife_array==NULL){
		printf("Could not allocate host data memory in %s!\n",argv[0]);
		return -314;
	}
	// read data into arrays
	for(i=0;i<npts;i++){
		fscanf(ipt_fptr,"%f,%f\n",&h_x[i],&h_y[i]);
	}
	fclose(ipt_fptr);
	
	// malloc on device
	hipMalloc((void**) &d_x, npts * sizeof(*d_x));
	checkCUDAError("hipMalloc d_x");
	hipMalloc((void**) &d_y, npts * sizeof(*d_y));
	checkCUDAError("hipMalloc d_y");
	hipMalloc((void**) &states, num_blocks * sizeof(hiprandState_t));
	checkCUDAError("hipMalloc states");
	hipMalloc((void**) &d_slope, Nbs * sizeof(*d_slope));
	checkCUDAError("hipMalloc d_slope");

	// store data
	hipMemcpy(d_x,h_x,npts*sizeof(*h_x),hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy d_x to device");
	hipMemcpy(d_y,h_y,npts*sizeof(*h_y),hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy d_y to device"); 
	
	if(Nbs==(1<<20)){ // we perform MC-BS
		// initialize random number generator on all blocks
		// start timing 
		hipEventRecord(start, 0);
		init_rand_kernel<<<num_blocks,thds_per_block>>>(time(0),states);
		// block until the device has completed
		hipDeviceSynchronize();
		//calculate elapsed time:
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		//Computes the elapsed time between two events (in milliseconds)
		hipEventElapsedTime(&exectime, start, stop);
		checkCUDAError("cuda_init_rand");
		printf("CUDA: init_rand time: %.5e\n",exectime*1e-3);
		
		// run the bootstrap
		// start timing 
		hipEventRecord(start, 0);
		mc_bs_slope_kernel<<<num_blocks,thds_per_block>>>(states, d_x, d_y, npts, Nbs, d_slope);
		// block until the device has completed
		hipDeviceSynchronize();
		//calculate elapsed time:
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		//Computes the elapsed time between two events (in milliseconds)
		hipEventElapsedTime(&exectime, start, stop);
		printf("CUDA: mc_bs_slope time: %.5e\n",exectime*1e-3);
		checkCUDAError("mc_bs_slope_kernel");
	}else{ // we perform full BS
		// run the bootstrap
		// start timing 
		hipEventRecord(start, 0);
		full_bs_slope_kernel<<<num_blocks,thds_per_block>>>(d_x, d_y, npts, Nbs, d_slope);
		// block until the device has completed
		hipDeviceSynchronize();
		//calculate elapsed time:
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		//Computes the elapsed time between two events (in milliseconds)
		hipEventElapsedTime(&exectime, start, stop);
		printf("CUDA: full bs_slope time: %.5e\n",exectime*1e-3);
		checkCUDAError("full bs_slope_kernel");
	}

	// transfer results to host
	// start timing 
	hipEventRecord(start, 0);
	// copy device memory to host
	hipMemcpy(h_slope,d_slope,Nbs*sizeof(*d_slope),hipMemcpyDeviceToHost);
	//calculate elapsed time:
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	checkCUDAError("hipMemcpy d_slope,d_s to host");

	printf("CUDA: cudaMemcpy_bs time: %.5e\n",exectime*1e-3);
	
	/* evaluate SLOPE bootstrap results */
	printf("--- Slope ---\n");
	// BCa percentile procedure
	// create jack-knife array of estimates for the slope
	jack_knife(h_x,h_y,npts,jack_knife_array,jack_knife_wrapper_slope);
	// calculate ahat
	ahat=my_ahat(jack_knife_array,npts);
	// sort to find median (also makes finding p_bias easy)
	middle=my_median(h_slope,Nbs,0);
	// calculate slope to find p_bias
	calc_BLUE_slope_intercept(h_x,h_y,npts,&slope,&intercept);
	printf("slope: %f, min: %f\n",slope,h_slope[0]);
	// find #{theta*<theta}
	for(i=0;i<Nbs && h_slope[i]<slope;i++);
	// this is the probability used to find z0 from std normal
	printf("#{theta*<theta}: %d\n",i);
	p_bias = (float)i/Nbs;
	printf("p_bias: %f\n", p_bias);
	z0=gsl_cdf_ugaussian_Pinv(p_bias);
	printf("z0: %f\n", z0);
	// lower and upper x-values associated with alpha/2 area 
	//     under left and right tails (respectively) of std normal
	z_lower=gsl_cdf_ugaussian_Pinv(alpha*0.5);
	z_upper=gsl_cdf_ugaussian_Qinv(alpha*0.5);
	printf("zinvs: (%f,%f)\n", z_lower,z_upper);
	// finally, these are the alpha values associated with the BCa percentiles
	BCa_alpha1=gsl_cdf_ugaussian_P(z0+(z0+z_lower)/(1-ahat*(z0+z_lower)));
	BCa_alpha2=gsl_cdf_ugaussian_P(z0+(z0+z_upper)/(1-ahat*(z0+z_upper)));
	printf("BCa_quantiles: (%f,%f)\n", BCa_alpha1,BCa_alpha2);
	// calculate BCa confidence intervals
	lower_BCa=h_slope[(int)(BCa_alpha1*Nbs)];
	upper_BCa=h_slope[(int)(BCa_alpha2*Nbs)];	
	// make histogram
	bin_width=(h_slope[Nbs-1]-h_slope[0])/hist_bins;
	printf("bin size: %.7e (%.7e-%.7e)/%d\n",bin_width,h_slope[Nbs-1],h_slope[0],hist_bins);
	for(bin=0,i=0;bin<hist_bins;bin++){
		hist_max[bin]=h_slope[0]+bin_width*(bin+1);
		//printf("bin max: %.7e, slope[%d]: %e\n",hist_max[bin],i,h_slope[i]);
		for(hist_counts[bin]=0;h_slope[i]<=hist_max[bin] && i<Nbs;i++,hist_counts[bin]++);
	}
	
	// non-bias-corrected percentiles
	lower_percentile=h_slope[(int)((alpha*0.5)*Nbs)];
	upper_percentile=h_slope[(int)((1.0-alpha*0.5)*Nbs)];
	
	// SE(median)
	SE_median=0;
	for(i=0;i<Nbs;i++){
		tmp=h_slope[i]-middle;
		SE_median+=tmp*tmp;
	}
	SE_median/=(Nbs-1);
	SE_median=sqrt(SE_median);
	SE_lower=middle-SE_median;
	SE_upper=middle+SE_median;
		
	// write percentile data to file
	sprintf(opt_fname,"%s-slope-CI.dat",ipt_fname);
	opt_fptr=fopen(opt_fname,"w");
	if(opt_fptr==NULL){
		printf("Could not open %s for writing!\n",opt_fname);
		return -1414;
	}
	mean=my_mean(h_slope,Nbs);
	fprintf(opt_fptr,"%.7e,%.7e,%.7e,%.7e,%.7e,%.7e,%.7e,%.7e,%d,%d\n",
			lower_percentile,upper_percentile,lower_BCa,upper_BCa,
			SE_lower,SE_upper,middle,mean,npts,Nbs);
	fclose(opt_fptr);
	
	// write histogram data to file
	sprintf(opt_fname,"%s-slope-histogram.dat",ipt_fname);
	opt_fptr=fopen(opt_fname,"w");
	if(opt_fptr==NULL){
		printf("Could not open %s for writing!\n",opt_fname);
		return -1414;
	}
	for(bin=0;bin<hist_bins;bin++){
		fprintf(opt_fptr,"%.7e,%d\n",hist_max[bin],hist_counts[bin]);
	}
	fclose(opt_fptr);
	
	printf("percentile:\t(%.7e,%.7e)\n BCa\t\t(%.7e,%.7e)\nMedian: %.7e, Mean: %.7e\n",
	        lower_percentile,upper_percentile,lower_BCa,upper_BCa,middle,mean);
	
		
	free(h_slope);
	free(h_x);
	free(h_y);
	free(jack_knife_array);
	
	hipFree(d_slope);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(states);	
	
    return 0;
}

float my_ahat(float xin[], int n){
	// calculate ahat for BCa
	// skewness estimator
	float top=0;
	float bot=0;
	float ahat;
	int i;
	float meen=my_mean(xin,n);
	float tmp,tmp1;
	
	for(i=0;i<n;i++){
		tmp=meen-xin[i];
		tmp1=tmp*tmp;
		bot+=tmp1;
		top+=tmp1*tmp;
	}
	bot=bot*bot*bot;
	bot=6*sqrt(bot);
	ahat=top/bot;
	
	return ahat;
}

void jack_knife(float xin[], float yin[], int n, float jack_theta[], 
	       float (*func)(float *, float *, int)){
	int i,j;
	float popA[n],popB[n];
	for(i=0;i<n;i++){
		for(j=0;j<n-1;j++){
			if(j!=i){// so long as we aren't equal, set populations
				popA[j]=xin[j];
				popB[j]=yin[j];
			}
		}
		popA[i]=xin[n-1];
		popB[i]=yin[n-1];
		// calculate func for this iteration
		jack_theta[i]=func(popA,popB,n-1);
	}
}

float jack_knife_wrapper_slope(float xin[], float yin[], int n){
	float discard,returnval;
	calc_BLUE_slope_intercept(xin,yin,n,&returnval,&discard);
	return returnval;
}

__host__ __device__ void calc_BLUE_slope_intercept(float xin[], float yin[], int n, 
			      float *slope_out, float *intercept_out){
	int i;
	float Sx,Sy,Sx2,Sxy;
	Sx=0; Sy=0;
	Sx2=0; Sxy=0;
	for(i=0;i<n;i++){
		Sx +=xin[i];
		Sy +=yin[i];
		Sx2+=xin[i]*xin[i];
		Sxy+=xin[i]*yin[i];
	}
	(*slope_out)=(n*Sxy-Sx*Sy)/(n*Sx2-Sx*Sx);
	(*intercept_out)=(Sy/n)-(*slope_out)*(Sx/n);
}

void checkCUDAError(const char *msg){
    hipError_t err = hipGetLastError();
    if( hipSuccess != err){
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
